#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <hip/hip_runtime.h>

#define SET_HIGH_BIT(number, size) (number | (unsigned int)(1 << (size*8-1)))
#define CLEAR_HIGH_BIT(number, size) (number & (~(unsigned int)(1 << (size*8-1))))
#define CLEAR_LOW_BIT(number) (number & (~1))
#define WAIT_GLOBAL(number, size, condition) while((__ldg(&number) >> (size*8-1)) != condition)
#define WAIT_SHARED(number, size, condition) while(((volatile uint32_t&)number >> (size*8-1)) != condition) {}
#define grid_size 1
#define block_size 2
#define thread_size 2

/* function: bit_stream << shift_count
 * size: the size of uint32_t
 * shift_count: shift number
 * shift_global: shift in and shift out bit
 */
__global__ void shift_left(uint32_t *bit_stream, int size, int shift_count, uint32_t* shift_global){
    printf("shift_count:%d\n", shift_count);
    __shared__ uint32_t shift_block[thread_size];
    int thread_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int bit_length = size * 8;

    // Todo: multi parameter will print error data.
    // printf("thread[%u,%u,(%u, %u, %u)]: bit_stream[%d] = %u\n", blockIdx.x, threadIdx.x,
    //     blockDim.x, blockDim.y, blockDim.z, thread_idx, bit_stream[thread_idx]);
    if(threadIdx.x == 0 && blockIdx.x == 0){
        //device inter
        shift_block[0] |= shift_global[0];
        __threadfence();
        shift_global[0] = CLEAR_HIGH_BIT(shift_global[0], sizeof(uint32_t));
    }

    if(threadIdx.x < blockDim.x-1){
        //block inner
        shift_block[threadIdx.x+1] = bit_stream[thread_idx] >> (bit_length - shift_count);
        __threadfence_block();
        shift_block[threadIdx.x+1] = SET_HIGH_BIT(shift_block[threadIdx.x+1], sizeof(uint32_t));
    }
    else{
        //block inter & device inter
        int block_idx = (blockIdx.x + 1) % gridDim.x;
        WAIT_GLOBAL(shift_global[block_idx], sizeof(uint32_t), 0);
        shift_global[block_idx] |= bit_stream[thread_idx] >> (bit_length - shift_count);
        __threadfence();
        shift_global[block_idx] = SET_HIGH_BIT(shift_global[block_idx], sizeof(uint32_t));
    }

    if(threadIdx.x == 0 && blockIdx.x != 0){
        //block inter
        WAIT_GLOBAL(shift_global[blockIdx.x], sizeof(uint32_t), 1);
        shift_block[0] = shift_global[blockIdx.x];
        shift_global[blockIdx.x] = CLEAR_HIGH_BIT(shift_global[blockIdx.x], sizeof(uint32_t));
    }

    WAIT_SHARED(shift_block[threadIdx.x], sizeof(uint32_t), 1);
    shift_block[threadIdx.x] = CLEAR_HIGH_BIT(shift_block[threadIdx.x], sizeof(uint32_t));
    bit_stream[thread_idx] = (bit_stream[thread_idx] << shift_count) | shift_block[threadIdx.x];

}

void get_bit_stream(uint32_t *bit_stream, int size){
    for(int i = 0; i < size; i++){
        bit_stream[i] = SET_HIGH_BIT(bit_stream[i], sizeof(uint32_t));
    }
}

int main()
{
    // Todo: iteration_time declared in the position won't be used in next for loop as the value is changed
    //  int iteration_time = grid_size;
    int grid_dim = block_size * thread_size;
    int bit_stream_size = grid_size * block_size * thread_size;

    uint32_t *bit_stream_gpu;
    uint32_t shift_global[block_size] = {0};
    uint32_t *shift_global_gpu;


    hipMalloc((void **)&bit_stream_gpu, sizeof(uint32_t)*grid_dim);
    hipMalloc((void **)&shift_global_gpu, sizeof(uint32_t)*block_size);

    printf("host_shift_global:%u\n", shift_global[0]);
    shift_global[0] = SET_HIGH_BIT(shift_global[0], sizeof(uint32_t));
    printf("host_shift_global[0]:%u\n", shift_global[0]);
    hipMemcpy(shift_global_gpu, shift_global, sizeof(uint32_t)*block_size, hipMemcpyHostToDevice);
    //Todo: get_bit_stream(bit_stream, bit_stream_size);

    int shift_count = 1;
    uint32_t bit_stream[bit_stream_size] = {0, 1, 2, 4, 8, 16, 32, 64};

    for(int i = 0; i < grid_size; i++){
        printf("iteration:%d", i);

        hipMemcpy(bit_stream_gpu, bit_stream+i*grid_dim,
            grid_dim*sizeof(uint32_t), hipMemcpyHostToDevice);

        dim3 threadsPerBlock(thread_size,1,1);
        shift_left<<<block_size, threadsPerBlock>>>(bit_stream_gpu, sizeof(uint32_t), shift_count, shift_global_gpu);

        hipMemcpy(bit_stream+i*grid_dim, bit_stream_gpu,
            grid_dim*sizeof(uint32_t), hipMemcpyDeviceToHost);
    }
    for (int j = 0; j < 4; j++){
        printf("host_bit_stream:%u\n",bit_stream[j]);
    }

    // Todo: when creating a new array, the value of bit_stream will be changed
    // uint32_t bit_stream_shift[bit_stream_size] = {0, 2, 4, 8, 16, 32, 64, 128};
    for (int j = 0; j < 4; j++){
        printf("host_bit_stream:%u\n",bit_stream[j]);
    }
    // for(int i = 0; i < bit_stream_size; i++){
    //     if(CLEAR_LOW_BIT(bit_stream[i]) == bit_stream_shift[i]){
    //         printf("[%d]: {bit_stream:%u, shift_global_stream:%u}\n", i,
    //             bit_stream[i] >> shift_count, bit_stream_shift[i]);
    //     }
    //     else{
    //         printf("ERROR[%d]: {bit_stream:%u, shift_global_stream:%u}\n", i,
    //             bit_stream[i] >> shift_count, bit_stream_shift[i]);
    //     }
    // }

    hipFree(shift_global_gpu);
    hipFree(bit_stream_gpu);

    return 0;
}
