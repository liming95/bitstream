#include <stdio.h>
#include <stdint.h>
#include <assert.h>
#include <hip/hip_runtime.h>

void bit_stream_add(uint32_t *stream1, uint32_t *stream2, uint32_t *result, size_t size){
    uint32_t carry = 0;
    for (size_t i = 0; i < size; i++) {
        __uint64_t sum = (__uint64_t)stream1[i] + stream2[i] + carry;
        result[i] = (uint32_t)sum;
        carry = (uint32_t)(sum >> 32);
    }
    if (carry) {
        result[size] = carry;
    }
}

/* Kogge-Stone:
 * 1.Generate carry (G) and propagate carry (P)
 * G(i) = A(i)B(i)
 * P(i) = A(i)^B(i)
 * 2. carry bit
 * C(i) = C(i-1) + P(i-1)C(i-1)
 * S(i) = P(i)^C(i)
*/
__global__ void bit_stream_add1_parallel(uint32_t *stream1, uint32_t *stream2,
    uint32_t *result, size_t size){

}

#define GRID_NUM 1
#define BLOCK_NUM 2
#define THREAD_NUM_PER_BLOCK 2
#define BYTE_SIZE 8
#define CLEAR_LOWEST_BIT_MASK 0xFFFFFFFE
#define LEFT_SHIFT_BIT(num, count) (num << count)
#define RIGHT_SHIFT_BIT(num, count) ((num >> count) & 0x00000001)
#define PROPAGATE_MASK(N) ((N) >= 32 ? 0xFFFFFFFFU : ((1U << (N)) - 1U))
#define SET(num, index) atomicOr(&num, 1 << index)
#define UNSET(num, index) atomicAnd(&num, ~(1 << index))
#define CHECK_LOCK(num, index) (num & (1 << index))
#define WAIT(num, index, target) while((CHECK_LOCK((volatile uint32_t&)num, index) >> index) != target)
#define WAIT_GLOBAL(num, count, target) while(((volatile uint32_t&)num & ((1 << count) - 1)) != target) \
{ printf("wait:"); printf("(num:%u",num); printf("count:%u", count); printf("%u\n)", target);}

__device__ uint32_t carry_global, propagate_global, carry_global_flag;
// __device__ int carry_global_lock, propagate_global_lock, carry_global_flag_lock;
// __device__ void lock(int *mutex) {
//     printf("muxte:%d\n",*mutex);
//     while (atomicCAS(mutex, 0, 1) != 0){ printf("muxte_while:%d\n",*mutex);};
//     printf("muxte2:%d\n",*mutex);

// }
// __device__ void unlock(int *mutex) {
//     atomicExch(mutex, 0);
// }

__device__ uint32_t get_carry_bit(uint32_t op1, uint32_t op2, uint32_t sum, int shift_count){
    uint32_t cond1, cond2;
    uint32_t op1_highest_bit = RIGHT_SHIFT_BIT(op1, shift_count);
    uint32_t op2_highest_bit = RIGHT_SHIFT_BIT(op2, shift_count);
    uint32_t sum_highest_bit = RIGHT_SHIFT_BIT(sum, shift_count);

    cond1 = op1_highest_bit & op2_highest_bit;
    cond2 = (op1_highest_bit ^ op2_highest_bit) & (!sum_highest_bit);
    return cond1 | cond2;
}
/*MatchStar:
 *
 */
__global__ void bit_stream_add2_parallel(uint32_t *stream1, uint32_t *stream2,
    uint32_t * result, size_t size){
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t bid = blockIdx.x;
    __shared__ uint32_t carry_block;
    __shared__ uint32_t propagate_block;
    uint32_t propagate_bit;
    result[tid] = stream1[tid] + stream2[tid];

    if (threadIdx.x == 0) {
        carry_block = 0;
        propagate_block = 0;
    }
    __syncthreads();

    //generate the G_bit and P_bit
    //printf("propa_lock:%d\n", propagate_lock);
    propagate_bit = result[tid] == (uint32_t)0xFFFFFFFF ? LEFT_SHIFT_BIT(1, threadIdx.x) : 0;
    atomicOr(&propagate_block, propagate_bit);
    //printf("propa_lock2:%d\n", propagate_lock);

    uint32_t carry_bit = 0;
    int shift_count = BYTE_SIZE * sizeof(uint32_t) - 1;
    carry_bit = get_carry_bit(stream1[tid], stream2[tid], result[tid], shift_count);
    __syncthreads();
    // Todo:the carry bit between block should be informed.
    printf("carry_bit:%u\n", carry_bit);
    // global
    if(bid == 0 & tid == 0){
        assert(CHECK_LOCK(carry_global_flag, 0) != 0);
        atomicOr(&carry_block, carry_global);
        atomicAnd(&carry_global, CLEAR_LOWEST_BIT_MASK);
        UNSET(carry_global_flag, 0);
        printf("carry_block_0_0: %u\n", carry_block);
    }
    // block
    if(threadIdx.x < blockDim.x-1){
        atomicOr(&carry_block, LEFT_SHIFT_BIT(carry_bit, threadIdx.x + 1));
        printf("carry_block_%u_%u : %u\n", bid, threadIdx.x, carry_block);
    }
    // global
    if(threadIdx.x == (uint32_t)(blockDim.x-1)){
        WAIT(carry_global_flag, (bid+1)%gridDim.x, 0);
        atomicOr(&carry_global, LEFT_SHIFT_BIT(carry_bit, (bid+1)%gridDim.x));
        SET(carry_global_flag, (bid+1)%gridDim.x);
    }

    //move carry global to block
    if(bid > 0 && threadIdx.x == 0){
        WAIT(carry_global_flag, bid, (uint32_t)1);
        atomicOr(&carry_block, RIGHT_SHIFT_BIT(carry_global, bid));
        atomicAnd(&carry_global, ~((uint32_t)LEFT_SHIFT_BIT(1, bid)));
        UNSET(carry_global_flag, bid);
        printf("carry_block_1-_0: %u\n", carry_block);
    }
    __syncthreads();

    if((bid == 0) && (threadIdx.x == 0)){
        WAIT(carry_global_flag, bid, (uint32_t)1);
        UNSET(carry_global_flag, bid);
    }

    //1. update carry_block
    if(threadIdx.x == 0){
        //pre-matchstar
        printf("carry_block_pre:%u\n", carry_block);
        uint32_t tmp1 = carry_block & propagate_block;

        //G_bit + P_bit
        tmp1 = tmp1 + propagate_block;
        propagate_bit = tmp1 == (uint32_t)PROPAGATE_MASK(gridDim.x) ? LEFT_SHIFT_BIT(1, bid) : 0;
        atomicOr(&propagate_global, propagate_bit);

        // set propagate_global
        shift_count = blockDim.x - 1;
        carry_bit = get_carry_bit(carry_block, propagate_block, tmp1, shift_count);

        WAIT(carry_global_flag, (bid+1)%gridDim.x, 0);
        atomicOr(&carry_global, LEFT_SHIFT_BIT(carry_bit, (bid+1)%gridDim.x));
        SET(carry_global_flag, (bid+1)%gridDim.x);
        printf("carry_global_flag:%u\n", carry_global_flag);

        // matchstar:update carry_global
        WAIT_GLOBAL(carry_global_flag, gridDim.x, (1<<gridDim.x)-1);
        uint32_t tmp2 = ((carry_global >> 1) << 1) & propagate_global;
        tmp2 += propagate_global;

        shift_count = gridDim.x -1;
        carry_bit = get_carry_bit(carry_global, propagate_global, tmp2, shift_count);

        tmp2 ^= propagate_global;
        tmp2 |= (carry_global >> 1) << 1;

        atomicOr(&carry_global, LEFT_SHIFT_BIT(carry_bit, 0));
        // correct the tmp1 except block 0
        if(bid > 0){
            tmp1 += RIGHT_SHIFT_BIT(tmp2, bid);
        }
        // update the carry_block: post-matchstar
        tmp1 ^= propagate_block;
        carry_block |= tmp1;
        printf("carry_block:%u\n", carry_block);
    }
    __syncthreads();
    //2.correct the result[tid]
    result[tid] += RIGHT_SHIFT_BIT(carry_block, threadIdx.x);
}

void fill_array(uint32_t* stream, int size){
    for(int i = 0; i < size; i++){
        stream[i] = i;
        stream[i] |= (uint32_t)(1 << 31);
    }
}

int main(){
    uint32_t *stream1, *stream2, *result;
    uint32_t *d_stream1, * d_stream2, *d_result;

    // alloc space and initialize array
    int size = GRID_NUM * BLOCK_NUM * THREAD_NUM_PER_BLOCK;
    stream1 = (uint32_t *)malloc(size * sizeof(uint32_t));
    fill_array(stream1, size);
    stream2 = (uint32_t *)malloc(size * sizeof(uint32_t));
    fill_array(stream2, size);
    result = (uint32_t *)malloc((size+1) * sizeof(uint32_t));

    // copy data from host to device
    int d_size = BLOCK_NUM * THREAD_NUM_PER_BLOCK;
    hipMalloc((void **)&d_stream1, d_size*sizeof(uint32_t));
    hipMalloc((void **)&d_stream2, d_size*sizeof(uint32_t));
    hipMalloc((void **)&d_result, d_size*sizeof(uint32_t));

    uint32_t initial_value = 0;
    hipMemcpyToSymbol(HIP_SYMBOL(carry_global), &initial_value, sizeof(uint32_t));
    hipMemcpyToSymbol(HIP_SYMBOL(propagate_global), &initial_value, sizeof(uint32_t));
    // cudaMemcpyToSymbol(carry_global_lock, &initial_value, sizeof(int));
    // cudaMemcpyToSymbol(propagate_global_lock, &initial_value, sizeof(int));
    // cudaMemcpyToSymbol(carry_global_flag_lock, &initial_value, sizeof(int));

    for(int i = 0; i < GRID_NUM; i++){
        initial_value = 1;
        hipMemcpyToSymbol(HIP_SYMBOL(carry_global_flag), &initial_value, sizeof(uint32_t));
        hipMemcpy(d_stream1, stream1, d_size*sizeof(uint32_t), hipMemcpyHostToDevice);
        hipMemcpy(d_stream2, stream2, d_size*sizeof(uint32_t), hipMemcpyHostToDevice);

        bit_stream_add2_parallel<<<BLOCK_NUM, THREAD_NUM_PER_BLOCK>>>(d_stream1, d_stream2,
            d_result, d_size);

        hipMemcpy(result+i*d_size, d_result, d_size*sizeof(uint32_t), hipMemcpyDeviceToHost);
        hipMemcpyFromSymbol(&initial_value, HIP_SYMBOL(carry_global_flag), sizeof(uint32_t));
        assert((initial_value & 0x01) != 0);
        initial_value = 0;
    }
    hipMemcpyFromSymbol(&result[size], HIP_SYMBOL(carry_global), sizeof(uint32_t));
    result[size] = RIGHT_SHIFT_BIT(result[size], 0);

    for(int i = 0; i < size; i++){
        printf("[%d]: %u + %u = %u\n", i, stream1[i], stream2[i], result[i]);
    }
    printf("carry:%u\n", result[size]);
    return 0;
}
